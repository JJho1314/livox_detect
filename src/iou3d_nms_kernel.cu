#include "hip/hip_runtime.h"
/*
3D IoU Calculation and Rotated NMS(modified from 2D NMS written by others)
Written by Shaoshuai Shi
All Rights Reserved 2019-2020.
*/

#include <iostream>
#include <stdio.h>
#include <vector>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/gather.h>
#include <thrust/transform.h>
#include <thrust/count.h>
#include <config.h>

#define THREADS_PER_BLOCK 16
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))
// int THREADS_PER_BLOCK_NMS =  sizeof(unsigned long long) * 8
// #define DEBUG
const float EPS = 1e-8;

struct Point
{
  float x, y;
  __device__ Point() {}
  __device__ Point(double _x, double _y)
  {
    x = _x, y = _y;
  }

  __device__ void set(float _x, float _y)
  {
    x = _x;
    y = _y;
  }

  __device__ Point operator+(const Point &b) const
  {
    return Point(x + b.x, y + b.y);
  }

  __device__ Point operator-(const Point &b) const
  {
    return Point(x - b.x, y - b.y);
  }
};

__device__ inline float cross(const Point &a, const Point &b)
{
  return a.x * b.y - a.y * b.x;
}

__device__ inline float cross(const Point &p1, const Point &p2, const Point &p0)
{
  return (p1.x - p0.x) * (p2.y - p0.y) - (p2.x - p0.x) * (p1.y - p0.y);
}

__device__ int check_rect_cross(const Point &p1, const Point &p2, const Point &q1, const Point &q2)
{
  int ret = min(p1.x, p2.x) <= max(q1.x, q2.x) &&
            min(q1.x, q2.x) <= max(p1.x, p2.x) &&
            min(p1.y, p2.y) <= max(q1.y, q2.y) &&
            min(q1.y, q2.y) <= max(p1.y, p2.y);
  return ret;
}

__device__ inline int check_in_box2d(const float *box, const Point &p)
{
  // params: (7) [x, y, z, dx, dy, dz, heading]
  const float MARGIN = 1e-2;

  float center_x = box[0], center_y = box[1];
  float angle_cos = cos(-box[6]), angle_sin = sin(-box[6]); // rotate the point in the opposite direction of box
  float rot_x = (p.x - center_x) * angle_cos + (p.y - center_y) * (-angle_sin);
  float rot_y = (p.x - center_x) * angle_sin + (p.y - center_y) * angle_cos;

  return (fabs(rot_x) < box[3] / 2 + MARGIN && fabs(rot_y) < box[4] / 2 + MARGIN);
}

__device__ inline int intersection(const Point &p1, const Point &p0, const Point &q1, const Point &q0, Point &ans)
{
  // fast exclusion
  if (check_rect_cross(p0, p1, q0, q1) == 0)
    return 0;

  // check cross standing
  float s1 = cross(q0, p1, p0);
  float s2 = cross(p1, q1, p0);
  float s3 = cross(p0, q1, q0);
  float s4 = cross(q1, p1, q0);

  if (!(s1 * s2 > 0 && s3 * s4 > 0))
    return 0;

  // calculate intersection of two lines
  float s5 = cross(q1, p1, p0);
  if (fabs(s5 - s1) > EPS)
  {
    ans.x = (s5 * q0.x - s1 * q1.x) / (s5 - s1);
    ans.y = (s5 * q0.y - s1 * q1.y) / (s5 - s1);
  }
  else
  {
    float a0 = p0.y - p1.y, b0 = p1.x - p0.x, c0 = p0.x * p1.y - p1.x * p0.y;
    float a1 = q0.y - q1.y, b1 = q1.x - q0.x, c1 = q0.x * q1.y - q1.x * q0.y;
    float D = a0 * b1 - a1 * b0;

    ans.x = (b0 * c1 - b1 * c0) / D;
    ans.y = (a1 * c0 - a0 * c1) / D;
  }

  return 1;
}

__device__ inline void rotate_around_center(const Point &center, const float angle_cos, const float angle_sin, Point &p)
{
  float new_x = (p.x - center.x) * angle_cos + (p.y - center.y) * (-angle_sin) + center.x;
  float new_y = (p.x - center.x) * angle_sin + (p.y - center.y) * angle_cos + center.y;
  p.set(new_x, new_y);
}

__device__ inline int point_cmp(const Point &a, const Point &b, const Point &center)
{
  return atan2(a.y - center.y, a.x - center.x) > atan2(b.y - center.y, b.x - center.x);
}

__device__ inline float box_overlap(const float *box_a, const float *box_b)
{
  // params box_a: [x, y, z, dx, dy, dz, heading]
  // params box_b: [x, y, z, dx, dy, dz, heading]

  float a_angle = box_a[6], b_angle = box_b[6];
  float a_dx_half = box_a[3] / 2, b_dx_half = box_b[3] / 2, a_dy_half = box_a[4] / 2, b_dy_half = box_b[4] / 2;
  float a_x1 = box_a[0] - a_dx_half, a_y1 = box_a[1] - a_dy_half;
  float a_x2 = box_a[0] + a_dx_half, a_y2 = box_a[1] + a_dy_half;
  float b_x1 = box_b[0] - b_dx_half, b_y1 = box_b[1] - b_dy_half;
  float b_x2 = box_b[0] + b_dx_half, b_y2 = box_b[1] + b_dy_half;

  Point center_a(box_a[0], box_a[1]);
  __device__ inline float iou_normal(float const *const a, float const *const b)
  {
    // params: a: [x, y, z, dx, dy, dz, heading]
    // params: b: [x, y, z, dx, dy, dz, heading]

    float left = fmaxf(a[0] - a[3] / 2, b[0] - b[3] / 2), right = fminf(a[0] + a[3] / 2, b[0] + b[3] / 2);
    float top = fmaxf(a[1] - a[4] / 2, b[1] - b[4] / 2), bottom = fminf(a[1] + a[4] / 2, b[1] + b[4] / 2);
    float width = fmaxf(right - left, 0.f), height = fmaxf(bottom - top, 0.f);
    float interS = width * height;
    float Sa = a[3] * a[4];
    float Sb = b[3] * b[4];
    return interS / fmaxf(Sa + Sb - interS, EPS);
  }
  Point center_b(box_b[0], box_b[1]);

#ifdef DEBUG
  printf("a: (%.3f, %.3f, %.3f, %.3f, %.3f), b: (%.3f, %.3f, %.3f, %.3f, %.3f)\n", a_x1, a_y1, a_x2, a_y2, a_angle,
         b_x1, b_y1, b_x2, b_y2, b_angle);
  printf("center a: (%.3f, %.3f), b: (%.3f, %.3f)\n", center_a.x, center_a.y, center_b.x, center_b.y);
#endif

  Point box_a_corners[5];
  box_a_corners[0].set(a_x1, a_y1);
  box_a_corners[1].set(a_x2, a_y1);
  box_a_corners[2].set(a_x2, a_y2);
  box_a_corners[3].set(a_x1, a_y2);

  Point box_b_corners[5];
  box_b_corners[0].set(b_x1, b_y1);
  box_b_corners[1].set(b_x2, b_y1);
  box_b_corners[2].set(b_x2, b_y2);
  box_b_corners[3].set(b_x1, b_y2);

  // get oriented corners
  float a_angle_cos = cos(a_angle), a_angle_sin = sin(a_angle);
  float b_angle_cos = cos(b_angle), b_angle_sin = sin(b_angle);

  for (int k = 0; k < 4; k++)
  {
#ifdef DEBUG
    printf("before corner %d: a(%.3f, %.3f), b(%.3f, %.3f) \n", k, box_a_corners[k].x, box_a_corners[k].y, box_b_corners[k].x, box_b_corners[k].y);
#endif
    rotate_around_center(center_a, a_angle_cos, a_angle_sin, box_a_corners[k]);
    rotate_around_center(center_b, b_angle_cos, b_angle_sin, box_b_corners[k]);
#ifdef DEBUG
    printf("corner %d: a(%.3f, %.3f), b(%.3f, %.3f) \n", k, box_a_corners[k].x, box_a_corners[k].y, box_b_corners[k].x, box_b_corners[k].y);
#endif
  }

  box_a_corners[4] = box_a_corners[0];
  box_b_corners[4] = box_b_corners[0];

  // get intersection of lines
  Point cross_points[16];
  Point poly_center;
  int cnt = 0, flag = 0;

  poly_center.set(0, 0);
  for (int i = 0; i < 4; i++)
  {
    for (int j = 0; j < 4; j++)
    {
      flag = intersection(box_a_corners[i + 1], box_a_corners[i], box_b_corners[j + 1], box_b_corners[j], cross_points[cnt]);
      if (flag)
      {
        poly_center = poly_center + cross_points[cnt];
        cnt++;
#ifdef DEBUG
        printf("Cross points (%.3f, %.3f): a(%.3f, %.3f)->(%.3f, %.3f), b(%.3f, %.3f)->(%.3f, %.3f) \n",
               cross_points[cnt - 1].x, cross_points[cnt - 1].y,
               box_a_corners[i].x, box_a_corners[i].y, box_a_corners[i + 1].x, box_a_corners[i + 1].y,
               box_b_corners[i].x, box_b_corners[i].y, box_b_corners[i + 1].x, box_b_corners[i + 1].y);
#endif
      }
    }
  }

  // check corners
  for (int k = 0; k < 4; k++)
  {
    if (check_in_box2d(box_a, box_b_corners[k]))
    {
      poly_center = poly_center + box_b_corners[k];
      cross_points[cnt] = box_b_corners[k];
      cnt++;
#ifdef DEBUG
      printf("b corners in a: corner_b(%.3f, %.3f)", cross_points[cnt - 1].x, cross_points[cnt - 1].y);
#endif
    }
    if (check_in_box2d(box_b, box_a_corners[k]))
    {
      poly_center = poly_center + box_a_corners[k];
      cross_points[cnt] = box_a_corners[k];
      cnt++;
#ifdef DEBUG
      printf("a corners in b: corner_a(%.3f, %.3f)", cross_points[cnt - 1].x, cross_points[cnt - 1].y);
#endif
    }
  }

  poly_center.x /= cnt;
  poly_center.y /= cnt;

  // sort the points of polygon
  Point temp;
  for (int j = 0; j < cnt - 1; j++)
  {
    for (int i = 0; i < cnt - j - 1; i++)
    {
      if (point_cmp(cross_points[i], cross_points[i + 1], poly_center))
      {
        temp = cross_points[i];
        cross_points[i] = cross_points[i + 1];
        cross_points[i + 1] = temp;
      }
    }
  }
  __device__ inline float iou_normal(float const *const a, float const *const b)
  {
    // params: a: [x, y, z, dx, dy, dz, heading]
    // params: b: [x, y, z, dx, dy, dz, heading]

    float left = fmaxf(a[0] - a[3] / 2, b[0] - b[3] / 2), right = fminf(a[0] + a[3] / 2, b[0] + b[3] / 2);
    float top = fmaxf(a[1] - a[4] / 2, b[1] - b[4] / 2), bottom = fminf(a[1] + a[4] / 2, b[1] + b[4] / 2);
    float width = fmaxf(right - left, 0.f), height = fmaxf(bottom - top, 0.f);
    float interS = width * height;
    float Sa = a[3] * a[4];
    float Sb = b[3] * b[4];
    return interS / fmaxf(Sa + Sb - interS, EPS);
  }
#ifdef DEBUG
  printf("cnt=%d\n", cnt);
  for (int i = 0; i < cnt; i++)
  {
    printf("All cross point %d: (%.3f, %.3f)\n", i, cross_points[i].x, cross_points[i].y);
  }
#endif

  // get the overlap areas
  float area = 0;
  for (int k = 0; k < cnt - 1; k++)
  {
    area += cross(cross_points[k] - cross_points[0], cross_points[k + 1] - cross_points[0]);
  }

  return fabs(area) / 2.0;
}

__device__ inline float iou_bev(const float *box_a, const float *box_b)
{
  // params box_a: [x, y, z, dx, dy, dz, heading]
  // params box_b: [x, y, z, dx, dy, dz, heading]
  float sa = box_a[3] * box_a[4];
  float sb = box_b[3] * box_b[4];
  float s_overlap = box_overlap(box_a, box_b);
  return s_overlap / fmaxf(sa + sb - s_overlap, EPS);
}

__global__ void boxes_overlap_kernel(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_overlap)
{
  // params boxes_a: (N, 7) [x, y, z, dx, dy, dz, heading]
  // params boxes_b: (M, 7) [x, y, z, dx, dy, dz, heading]
  const int a_idx = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
  const int b_idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

  if (a_idx >= num_a || b_idx >= num_b)
  {
    return;
  }
  const float *cur_box_a = boxes_a + a_idx * 7;
  const float *cur_box_b = boxes_b + b_idx * 7;
  float s_overlap = box_overlap(cur_box_a, cur_box_b);
  ans_overlap[a_idx * num_b + b_idx] = s_overlap;
}

__global__ void boxes_iou_bev_kernel(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_iou)
{
  // params boxes_a: (N, 7) [x, y, z, dx, dy, dz, heading]
  // params boxes_b: (M, 7) [x, y, z, dx, dy, dz, heading]
  const int a_idx = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
  const int b_idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

  if (a_idx >= num_a || b_idx >= num_b)
  {
    return;
  }

  const float *cur_box_a = boxes_a + a_idx * 7;
  const float *cur_box_b = boxes_b + b_idx * 7;
  float cur_iou_bev = iou_bev(cur_box_a, cur_box_b);
  ans_iou[a_idx * num_b + b_idx] = cur_iou_bev;
}

__global__ void nms_kernel(const int boxes_num, const float nms_overlap_thresh,
                           const float *boxes, unsigned long long *mask)
{
  // params: boxes (N, 7) [x, y, z, dx, dy, dz, heading]
  // params: mask (N, N/THREADS_PER_BLOCK_NMS)

  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size = fminf(boxes_num - row_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);
  const int col_size = fminf(boxes_num - col_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);

  __shared__ float block_boxes[THREADS_PER_BLOCK_NMS * 7];

  if (threadIdx.x < col_size)
  {
    block_boxes[threadIdx.x * 7 + 0] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 0];
    block_boxes[threadIdx.x * 7 + 1] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 1];
    block_boxes[threadIdx.x * 7 + 2] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 2];
    block_boxes[threadIdx.x * 7 + 3] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 3];
    block_boxes[threadIdx.x * 7 + 4] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 4];
    block_boxes[threadIdx.x * 7 + 5] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 5];
    block_boxes[threadIdx.x * 7 + 6] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 6];
  }
  __syncthreads();

  if (threadIdx.x < row_size)
  {
    const int cur_box_idx = THREADS_PER_BLOCK_NMS * row_start + threadIdx.x;
    const float *cur_box = boxes + cur_box_idx * 7;

    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start)
    {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++)
    {
      if (iou_bev(cur_box, block_boxes + i * 7) > nms_overlap_thresh)
      {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(boxes_num, THREADS_PER_BLOCK_NMS);
    mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

__global__ void tmpfunc(const int boxes_num, const float nms_overlap_thresh,
                        const float *reg, const float *height, const float *dim, const float *rot, const int *indexs, unsigned long long *mask, float *block_boxes)
{
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;
  const int row_size = fminf(boxes_num - row_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);
  const int col_size = fminf(boxes_num - col_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);

  if (row_start + col_start == 0 && threadIdx.x < col_size)
  {
    const int col_actual_idx = indexs[THREADS_PER_BLOCK_NMS * col_start + threadIdx.x];

    block_boxes[threadIdx.x * 7 + 0] = reg[col_actual_idx];
    block_boxes[threadIdx.x * 7 + 1] = reg[OUTPUT_H * OUTPUT_W + col_actual_idx];
    block_boxes[threadIdx.x * 7 + 2] = height[col_actual_idx];
    block_boxes[threadIdx.x * 7 + 3] = dim[col_actual_idx];
    block_boxes[threadIdx.x * 7 + 4] = dim[col_actual_idx + OUTPUT_W * OUTPUT_H];
    block_boxes[threadIdx.x * 7 + 5] = dim[col_actual_idx + OUTPUT_W * OUTPUT_H * 2];
    float theta = atan2f(rot[col_actual_idx], rot[col_actual_idx + OUTPUT_W * OUTPUT_H]);
    block_boxes[threadIdx.x * 7 + 6] = theta;
  }
}

__global__ void raw_nms_kernel(const int boxes_num, const float nms_overlap_thresh,
                               const float *reg, const float *height, const float *dim, const float *rot, const int *indexs, unsigned long long *mask)
{
  // params: mask (N, N/THREADS_PER_BLOCK_NMS)

  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;
  const int row_size = fminf(boxes_num - row_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);
  const int col_size = fminf(boxes_num - col_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);

  __shared__ float block_boxes[THREADS_PER_BLOCK_NMS * 7];

  if (threadIdx.x < col_size)
  {
    const int col_actual_idx = indexs[THREADS_PER_BLOCK_NMS * col_start + threadIdx.x];
    const int xIdx = col_actual_idx % OUTPUT_W;
    const int yIdx = col_actual_idx / OUTPUT_W;

    // encode boxs according kitti  format : (N, 7) [x, y, z, dy, dx, dz, heading]
    block_boxes[threadIdx.x * 7 + 0] = (reg[col_actual_idx] + xIdx) * OUT_SIZE_FACTOR * X_STEP + X_MIN;
    block_boxes[threadIdx.x * 7 + 1] = (reg[OUTPUT_H * OUTPUT_W + col_actual_idx] + yIdx) * OUT_SIZE_FACTOR * Y_STEP + Y_MIN;
    block_boxes[threadIdx.x * 7 + 2] = height[col_actual_idx];
    block_boxes[threadIdx.x * 7 + 4] = dim[col_actual_idx];
    block_boxes[threadIdx.x * 7 + 3] = dim[col_actual_idx + OUTPUT_W * OUTPUT_H];
    block_boxes[threadIdx.x * 7 + 5] = dim[col_actual_idx + OUTPUT_W * OUTPUT_H * 2];
    float theta = atan2f(rot[col_actual_idx], rot[col_actual_idx + OUTPUT_W * OUTPUT_H]);
    theta = -theta - 3.1415926 / 2;
    block_boxes[threadIdx.x * 7 + 6] = theta;
  }
  __syncthreads();

  if (threadIdx.x < row_size)
  {
    const int row_actual_idx = indexs[THREADS_PER_BLOCK_NMS * row_start + threadIdx.x];
    const int cur_box_idx = THREADS_PER_BLOCK_NMS * row_start + threadIdx.x;
    const int xIdx = row_actual_idx % OUTPUT_W;
    const int yIdx = row_actual_idx / OUTPUT_W;

    // encode boxs according kitti  format : (N, 7) [x, y, z, dy, dx, dz, heading]
    float cur_box[7];
    cur_box[0] = (reg[row_actual_idx] + xIdx) * OUT_SIZE_FACTOR * X_STEP + X_MIN;
    cur_box[1] = (reg[OUTPUT_H * OUTPUT_W + row_actual_idx] + yIdx) * OUT_SIZE_FACTOR * Y_STEP + Y_MIN;
    cur_box[2] = height[row_actual_idx];
    cur_box[4] = dim[row_actual_idx];
    cur_box[3] = dim[row_actual_idx + OUTPUT_W * OUTPUT_H];
    cur_box[5] = dim[row_actual_idx + OUTPUT_W * OUTPUT_H * 2];
    float theta = atan2f(rot[row_actual_idx], rot[row_actual_idx + OUTPUT_W * OUTPUT_H]);
    theta = -theta - 3.1415926 / 2;
    cur_box[6] = theta;

    // const float *cur_box = boxes + cur_box_idx * 7;

    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start)
    {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++)
    {
      if (iou_bev(cur_box, block_boxes + i * 7) > nms_overlap_thresh)
      {
        t |= 1ULL << i;
      }
    }

    const int col_blocks = DIVUP(boxes_num, THREADS_PER_BLOCK_NMS);
    // assume cur_box_idx = 21, col_start = 0, row_start = 0 , threadIdx = 21, mark 21 th box and top 64 boxes
    mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

__device__ inline float iou_normal(float const *const a, float const *const b)
{
  // params: a: [x, y, z, dx, dy, dz, heading]
  // params: b: [x, y, z, dx, dy, dz, heading]

  float left = fmaxf(a[0] - a[3] / 2, b[0] - b[3] / 2), right = fminf(a[0] + a[3] / 2, b[0] + b[3] / 2);
  float top = fmaxf(a[1] - a[4] / 2, b[1] - b[4] / 2), bottom = fminf(a[1] + a[4] / 2, b[1] + b[4] / 2);
  float width = fmaxf(right - left, 0.f), height = fmaxf(bottom - top, 0.f);
  float interS = width * height;
  float Sa = a[3] * a[4];
  float Sb = b[3] * b[4];
  return interS / fmaxf(Sa + Sb - interS, EPS);
}

/////////////////////////////////////////////////////////////////////////////////////BEGIN////////////////////////////////////////////////////////////////////////////////////////////

__global__ void boxAssignKernel(float *reg, float *height, float *dim, float *rot, float *boxes, float *score, int *label, float *out_score, int *out_label,
                                int *validIndexs, int output_h, int output_w)
{
  int boxId = blockIdx.x;
  int channel = threadIdx.x;
  int idx = validIndexs[boxId];
  if (channel == 0)
    boxes[boxId * 7 + 0] = reg[idx];
  else if (channel == 1)
    boxes[boxId * 7 + 1] = reg[idx + output_w * output_h];
  else if (channel == 2)
    boxes[boxId * 7 + 2] = height[idx];
  else if (channel == 3)
    boxes[boxId * 7 + 3] = dim[idx];
  else if (channel == 4)
    boxes[boxId * 7 + 4] = dim[idx + output_h * output_w];
  else if (channel == 5)
    boxes[boxId * 7 + 5] = dim[idx + 2 * output_w * output_h];
  else if (channel == 6)
  {
    float theta = atan2f(rot[0 * output_h * output_w + idx], rot[1 * output_h * output_w + idx]);
    theta = -theta - 3.1415926 / 2;
    boxes[boxId * 7 + 6] = theta;
  }
  // else if(channel == 7)
  // out_score[boxId] = score[idx];
  else if (channel == 8)
    out_label[boxId] = label[idx];
}
void boxAssignLauncher(float *reg, float *height, float *dim, float *rot, float *boxes, float *score, int *label, float *out_score, int *out_label,
                       int *validIndexs, int boxSize, int output_h, int output_w)
{
  boxAssignKernel<<<boxSize, 9>>>(reg, height, dim, rot, boxes, score, label, out_score, out_label, validIndexs, output_h, output_w);
}

__global__ void indexAssign(int *indexs)
{
  int yIdx = blockIdx.x;
  int xIdx = threadIdx.x;
  int idx = yIdx * blockDim.x + xIdx;
  indexs[idx] = idx;
}

void indexAssignLauncher(int *indexs, int output_h, int output_w)
{
  indexAssign<<<output_h, output_w>>>(indexs);
}

// compute how many scores are valid
struct is_greater
{
  is_greater(float thre) : _thre(thre) {}
  __host__ __device__ bool operator()(const float &x)
  {
    return x >= _thre;
  }
  float _thre;
};
struct is_odd
{
  __host__ __device__ bool operator()(const int &x)
  {
    return true;
  }
};

__global__ void findValidScoreNumKernel_(float *score, float *thre, float *N)
{
  int yIdx = blockIdx.x;
  int xIdx = threadIdx.x;
  int idx = yIdx * blockDim.x + xIdx;
  if (score[idx] >= 0.1)
    atomicAdd(N, 1.0);
}

int findValidScoreNum(float *score, float thre, int output_h, int output_w)
{
  // thrust::device_vector<float> score_vec(score,score + output_h * output_w);
  return thrust::count_if(thrust::device, score, score + output_h * output_w, is_greater(thre));
  // return thrust::count_if(thrust::device, score_vec.begin(),score_vec.end(),is_greater(thre));
}

// int findValidScoreNum(float* score, float thre, int output_h, int output_w ) { //,  thrust::host_vector<int> host_box_size) {
//     float box_size[1];
//     float *box_size_;
//     float* thre_;
//     hipMalloc((void**)&box_size_, 2*sizeof(float));
//     hipMemset(box_size_,0,sizeof(float));
//     hipMemset(box_size_+1,1,sizeof(float));
//     hipMalloc((void**)&thre_, sizeof(float));
//     hipMemset(thre_,0.1,sizeof(float));
//     std::cout <<"using atomic add \n";
//     findValidScoreNumKernel_<<<output_h,output_w >>>(score, thre_, box_size_);
//     hipMemcpy(box_size, box_size_, sizeof(float), hipMemcpyDeviceToHost);
//     std::cout << "valid score num " << box_size[0] << std::endl;
//     hipFree(box_size_);
//     hipFree(thre_);
//     return 194;
//     // thrust::device_vector<int> box_size_vec(box_size, box_size + 1);
//     // thrust::copy(box_size_vec.begin(), box_size_vec.end(),host_box_size.begin());
// }

void sort_by_key(float *keys, int *values, int size)
{

  thrust::sequence(thrust::device, values, values + size);
  // size = OUTPUT_H * OUTPUT_W;
  thrust::sort_by_key(thrust::device, keys, keys + size, values, thrust::greater<float>());
}

void gather_all(float *host_boxes, int *host_label,
                float *reg, float *height, float *dim, float *rot, float *sorted_score, int32_t *label,
                int *dev_indexs, long *host_keep_indexs, int boxSizeBef, int boxSizeAft)
{

  // copy keep_indexs from host to device
  // int* tmp_keep_indexs = static_cast<int*>(host_keep_indexs);
  thrust::device_vector<long> dev_keep_indexs(host_keep_indexs, host_keep_indexs + boxSizeAft);
  // thrust::host_vector<long> host_keep_indexs_vec(host_keep_indexs,host_keep_indexs+boxSizeAft);
  // // thrust::copy(host_keep_indexs,host_keep_indexs+boxSizeAft, dev_keep_indexs.begin());
  // thrust::copy(host_keep_indexs_vec.begin(), host_keep_indexs_vec.end(), dev_keep_indexs.begin());
  // gather keeped indexs after nms
  thrust::device_vector<int> dev_indexs_bef(dev_indexs, dev_indexs + boxSizeBef);
  thrust::device_vector<int> dev_indexs_aft(boxSizeAft);
  thrust::gather(dev_keep_indexs.begin(), dev_keep_indexs.end(),
                 dev_indexs_bef.begin(),
                 dev_indexs_aft.begin());
  // gather boxes, score, label
  thrust::device_vector<float> tmp_boxes(boxSizeAft * 9);
  thrust::device_vector<int> tmp_label(boxSizeAft);
  // gather x, y
  thrust::device_vector<float> reg_vec(reg, reg + OUTPUT_H * OUTPUT_W * 2);
  thrust::gather(dev_indexs_aft.begin(), dev_indexs_aft.end(), reg_vec.begin(), tmp_boxes.begin());
  thrust::gather(dev_indexs_aft.begin(), dev_indexs_aft.end(), reg_vec.begin() + OUTPUT_W * OUTPUT_H, tmp_boxes.begin() + boxSizeAft);
  // gather height
  thrust::device_vector<float> height_vec(height, height + OUTPUT_H * OUTPUT_W);
  thrust::gather(dev_indexs_aft.begin(), dev_indexs_aft.end(), height_vec.begin(), tmp_boxes.begin() + 2 * boxSizeAft);
  // gather  dim
  thrust::device_vector<float> dim_vec(dim, dim + 3 * OUTPUT_H * OUTPUT_W);
  thrust::gather(dev_indexs_aft.begin(), dev_indexs_aft.end(), dim_vec.begin() + OUTPUT_W * OUTPUT_H * 0, tmp_boxes.begin() + 3 * boxSizeAft);
  thrust::gather(dev_indexs_aft.begin(), dev_indexs_aft.end(), dim_vec.begin() + OUTPUT_W * OUTPUT_H * 1, tmp_boxes.begin() + 4 * boxSizeAft);
  thrust::gather(dev_indexs_aft.begin(), dev_indexs_aft.end(), dim_vec.begin() + OUTPUT_W * OUTPUT_H * 2, tmp_boxes.begin() + 5 * boxSizeAft);
  // gather rotation
  thrust::device_vector<float> rot_vec(rot, rot + 2 * OUTPUT_H * OUTPUT_W);
  thrust::gather(dev_indexs_aft.begin(), dev_indexs_aft.end(), rot_vec.begin() + OUTPUT_W * OUTPUT_H * 0, tmp_boxes.begin() + 6 * boxSizeAft);
  thrust::gather(dev_indexs_aft.begin(), dev_indexs_aft.end(), rot_vec.begin() + OUTPUT_W * OUTPUT_H * 1, tmp_boxes.begin() + 7 * boxSizeAft);
  // gather score
  thrust::device_vector<float> sorted_score_vec(sorted_score, sorted_score + 1 * OUTPUT_H * OUTPUT_W);
  thrust::gather(dev_keep_indexs.begin(), dev_keep_indexs.end(), sorted_score_vec.begin() + OUTPUT_W * OUTPUT_H * 0, tmp_boxes.begin() + 8 * boxSizeAft);
  // gather label
  thrust::device_vector<int> label_vec(label, label + 1 * OUTPUT_H * OUTPUT_W);
  thrust::gather(dev_indexs_aft.begin(), dev_indexs_aft.end(), label_vec.begin() + OUTPUT_W * OUTPUT_H * 0, tmp_label.begin());

  // copy values from device => host
  // host_boxes = tmp_boxes;
  // host_label = tmp_label;
  thrust::copy(tmp_boxes.begin(), tmp_boxes.end(), host_boxes);
  thrust::copy(tmp_label.begin(), tmp_label.end(), host_label);
}

///////////////////////////////////////////////////////////////////////////////////END//////////////////////////////////////////////////////////////////////////////////////////

__global__ void nms_normal_kernel(const int boxes_num, const float nms_overlap_thresh,
                                  const float *boxes, unsigned long long *mask)
{
  // params: boxes (N, 7) [x, y, z, dx, dy, dz, heading]
  // params: mask (N, N/THREADS_PER_BLOCK_NMS)

  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size = fminf(boxes_num - row_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);
  const int col_size = fminf(boxes_num - col_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);

  __shared__ float block_boxes[THREADS_PER_BLOCK_NMS * 7];

  if (threadIdx.x < col_size)
  {
    block_boxes[threadIdx.x * 7 + 0] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 0];
    block_boxes[threadIdx.x * 7 + 1] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 1];
    block_boxes[threadIdx.x * 7 + 2] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 2];
    block_boxes[threadIdx.x * 7 + 3] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 3];
    block_boxes[threadIdx.x * 7 + 4] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 4];
    block_boxes[threadIdx.x * 7 + 5] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 5];
    block_boxes[threadIdx.x * 7 + 6] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 6];
  }
  __syncthreads();

  if (threadIdx.x < row_size)
  {
    const int cur_box_idx = THREADS_PER_BLOCK_NMS * row_start + threadIdx.x;
    const float *cur_box = boxes + cur_box_idx * 7;

    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start)
    {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++)
    {
      if (iou_normal(cur_box, block_boxes + i * 7) > nms_overlap_thresh)
      {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(boxes_num, THREADS_PER_BLOCK_NMS);
    mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

void boxesoverlapLauncher(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_overlap)
{

  dim3 blocks(DIVUP(num_b, THREADS_PER_BLOCK), DIVUP(num_a, THREADS_PER_BLOCK)); // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

  boxes_overlap_kernel<<<blocks, threads>>>(num_a, boxes_a, num_b, boxes_b, ans_overlap);
#ifdef DEBUG
  hipDeviceSynchronize(); // for using printf in kernel function
#endif
}

void boxesioubevLauncher(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_iou)
{

  dim3 blocks(DIVUP(num_b, THREADS_PER_BLOCK), DIVUP(num_a, THREADS_PER_BLOCK)); // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

  boxes_iou_bev_kernel<<<blocks, threads>>>(num_a, boxes_a, num_b, boxes_b, ans_iou);
#ifdef DEBUG
  hipDeviceSynchronize(); // for using printf in kernel function
#endif
}

void nmsLauncher(const float *boxes, unsigned long long *mask, int boxes_num, float nms_overlap_thresh)
{
  dim3 blocks(DIVUP(boxes_num, THREADS_PER_BLOCK_NMS),
              DIVUP(boxes_num, THREADS_PER_BLOCK_NMS));
  dim3 threads(THREADS_PER_BLOCK_NMS);

  nms_kernel<<<blocks, threads>>>(boxes_num, nms_overlap_thresh, boxes, mask);
}

void nmsNormalLauncher(const float *boxes, unsigned long long *mask, int boxes_num, float nms_overlap_thresh)
{
  dim3 blocks(DIVUP(boxes_num, THREADS_PER_BLOCK_NMS),
              DIVUP(boxes_num, THREADS_PER_BLOCK_NMS));
  dim3 threads(THREADS_PER_BLOCK_NMS);
  nms_normal_kernel<<<blocks, threads>>>(boxes_num, nms_overlap_thresh, boxes, mask);
}

void rawNmsLauncher(const float *reg, const float *height, const float *dim, const float *rot, const int *indexs, unsigned long long *mask, int boxes_num, float nms_overlap_thresh)
{
  dim3 blocks(DIVUP(boxes_num, THREADS_PER_BLOCK_NMS),
              DIVUP(boxes_num, THREADS_PER_BLOCK_NMS));
  dim3 threads(THREADS_PER_BLOCK_NMS);
  raw_nms_kernel<<<blocks, threads>>>(boxes_num, nms_overlap_thresh, reg, height, dim, rot, indexs, mask);
}

int rawNmsGpu(const float *reg, const float *height, const float *dim, const float *rot,
              const int *indexs, long *host_keep_data, unsigned long long *mask_cpu, unsigned long long *remv_cpu,
              int boxes_num, float nms_overlap_thresh)
{
  // params boxes: (N, 7) [x, y, z, dx, dy, dz, heading]
  // params keep: (N)

  // int boxes_num = boxes.size(0);
  // const float * boxes_data = boxes.data<float>();
  // long * keep_data = keep.data<long>();

  const int col_blocks = DIVUP(boxes_num, THREADS_PER_BLOCK_NMS);

  unsigned long long *mask_data = NULL;
  hipMalloc((void **)&mask_data, boxes_num * col_blocks * sizeof(unsigned long long));
  rawNmsLauncher(reg, height, dim, rot, indexs, mask_data, boxes_num, nms_overlap_thresh);

  // unsigned long long mask_cpu[boxes_num * col_blocks];
  // unsigned long long *mask_cpu = new unsigned long long [boxes_num * col_blocks];
  // std::vector<unsigned long long> mask_cpu(boxes_num * col_blocks);

  //    printf("boxes_num=%d, col_blocks=%d\n", boxes_num, col_blocks);
  hipMemcpy(mask_cpu, mask_data, boxes_num * col_blocks * sizeof(unsigned long long),
             hipMemcpyDeviceToHost);

  // TODO : CUT HERE ! ! !
  hipFree(mask_data);

  // unsigned long long remv_cpu[col_blocks];
  // memset(remv_cpu, 0, col_blocks * sizeof(unsigned long long));

  memset(remv_cpu, 0, col_blocks * sizeof(unsigned long long));
  int num_to_keep = 0;

  for (int i = 0; i < boxes_num; i++)
  {
    int nblock = i / THREADS_PER_BLOCK_NMS;
    int inblock = i % THREADS_PER_BLOCK_NMS;

    if (!(remv_cpu[nblock] & (1ULL << inblock)))
    {
      host_keep_data[num_to_keep++] = i;
      for (int j = nblock; j < col_blocks; j++)
      {
        remv_cpu[j] |= mask_cpu[i * col_blocks + j];
      }
    }
  }

  if (hipSuccess != hipGetLastError())
    printf("Error!\n");
  return num_to_keep;
}
